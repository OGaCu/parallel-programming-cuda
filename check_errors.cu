#define wbCheck(stmt) do {\ 
    hipError_t err = stmt;\ 
    if(err != hipSuccess) { \
        wbLog(ERROR, "Failed to run stmt", #stmt); \
        wbLog(ERROR, "Got CUDA error ...", hipGetErrorString(err)); \
        return -1; \
        } \
    } while(0)

// example usage
int check_error = wbCheck(hipMalloc(...));